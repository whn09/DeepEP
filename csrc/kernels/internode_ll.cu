#include "hip/hip_runtime.h"
#include "configs.cuh"
#include "exception.cuh"
#include "launch.cuh"
#include "utils.cuh"
// #include "ibgda_device.cuh"
#include "nvshmem_device.cuh"
// #include "efa_device.cuh"

namespace deep_ep {

namespace internode_ll {

template <bool use_warp_sync = false>
__forceinline__ __device__ bool is_rank_masked(int* mask_buffer_ptr, int rank) {
    if (mask_buffer_ptr == nullptr) {
        return false;
    }
    if constexpr (use_warp_sync) {
        return __shfl_sync(0xffffffff, ld_acquire_global(mask_buffer_ptr + rank), 0) != 0;
    } else {
        return ld_acquire_global(mask_buffer_ptr + rank) != 0;
    }
}

template <int kNumThreads>
__forceinline__ __device__ void barrier(int thread_id, int rank, int num_ranks, int* mask_buffer_ptr, int* sync_buffer_ptr) {
    EP_DEVICE_ASSERT(kNumThreads >= num_ranks);

    // Quiet all QPs
    // auto qps_per_rank = ibgda_get_state()->num_rc_per_pe * ibgda_get_state()->num_devices_initialized;

    // for (int i = thread_id; i < qps_per_rank * (num_ranks - 1); i += kNumThreads) {
    //     auto dst_rank = (rank + 1 + i / qps_per_rank) % num_ranks;
    //     auto qp_id = i % qps_per_rank;
    //     nvshmemi_ibgda_quiet(dst_rank, qp_id);
    // }
    for (int i = thread_id; i < (num_ranks - 1); i += kNumThreads) {
        auto dst_rank = (rank + 1 + i) % num_ranks;
        // 对每个目标 rank 调用一次 quiet
        // if (i == thread_id) {  // 避免重复调用
        //     nvshmem_quiet();
        // }
        nvshmem_quiet();
    }

    // Update local counter
    if (thread_id == 0)
        atomicAdd(sync_buffer_ptr + rank, -1);
    __syncthreads();

    int cnt = sync_buffer_ptr[rank];
    // Update remote counter and wait for local counter to be updated
    if (thread_id < num_ranks && thread_id != rank) {
        const auto dst_rank = thread_id;
        const auto dst_ptr = reinterpret_cast<uint64_t>(sync_buffer_ptr + rank);
        const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);

        if (not is_rank_masked(mask_buffer_ptr, dst_rank)) {
            if (dst_p2p_ptr == 0) {
                nvshmemi_ibgda_rma_p(reinterpret_cast<int*>(dst_ptr), cnt, dst_rank, 0);
            } else {
                st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), cnt);
            }

            auto start_time = clock64();
            uint64_t wait_recv_cost = 0;
            while (ld_acquire_sys_global(sync_buffer_ptr + dst_rank) != cnt            // remote is not ready
                   && (wait_recv_cost = clock64() - start_time) <= NUM_TIMEOUT_CYCLES  // not timeout
            )
                ;
            // Mask rank if timeout
            if (wait_recv_cost > NUM_TIMEOUT_CYCLES) {
                printf("Warning: DeepEP timeout for barrier, rank %d, dst_rank %d\n", rank, dst_rank);
                if (mask_buffer_ptr == nullptr)
                    trap();
                atomicExch(mask_buffer_ptr + dst_rank, 1);
            }
        }
    }
    __syncthreads();
}

template <int kNumThreads>
__launch_bounds__(kNumThreads, 1) __global__ void clean_low_latency_buffer(int* clean_0,
                                                                           int num_clean_int_0,
                                                                           int* clean_1,
                                                                           int num_clean_int_1,
                                                                           int rank,
                                                                           int num_ranks,
                                                                           int* mask_buffer_ptr,
                                                                           int* sync_buffer_ptr) {
    auto thread_id = static_cast<int>(threadIdx.x);

    // Barrier before cleaning (in case of unfinished chunked EP)
    if (sync_buffer_ptr == nullptr)
        nvshmemx_barrier_all_block();
    else
        barrier<kNumThreads>(thread_id, rank, num_ranks, mask_buffer_ptr, sync_buffer_ptr);

    // Clean
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_0; i += kNumThreads)
        clean_0[i] = 0;
    #pragma unroll
    for (int i = thread_id; i < num_clean_int_1; i += kNumThreads)
        clean_1[i] = 0;

    // Barrier after cleaning (make sure the low-latency mode works fine)
    if (sync_buffer_ptr == nullptr)
        nvshmemx_barrier_all_block();
    else
        barrier<kNumThreads>(thread_id, rank, num_ranks, mask_buffer_ptr, sync_buffer_ptr);
}

void clean_low_latency_buffer(int* clean_0,
                              int num_clean_int_0,
                              int* clean_1,
                              int num_clean_int_1,
                              int rank,
                              int num_ranks,
                              int* mask_buffer_ptr,
                              int* sync_buffer_ptr,
                              hipStream_t stream) {
    constexpr int kNumThreads = 256;

    SETUP_LAUNCH_CONFIG(1, kNumThreads, stream);

    LAUNCH_KERNEL(&cfg,
                  clean_low_latency_buffer<kNumThreads>,
                  clean_0,
                  num_clean_int_0,
                  clean_1,
                  num_clean_int_1,
                  rank,
                  num_ranks,
                  mask_buffer_ptr,
                  sync_buffer_ptr);
}

template <bool kUseFP8, bool kUseUE8M0, int kHidden>
__global__ __launch_bounds__(1024, 1) void dispatch(void* packed_recv_x,
                                                    void* packed_recv_x_scales,
                                                    int* packed_recv_src_info,
                                                    int64_t* packed_recv_layout_range,
                                                    int* packed_recv_count,
                                                    int* mask_buffer_ptr,
                                                    int* cumulative_local_expert_recv_stats,
                                                    int64_t* dispatch_wait_recv_cost_stats,
                                                    void* rdma_recv_x,
                                                    int* rdma_recv_count,
                                                    void* rdma_x,
                                                    const void* x,
                                                    const topk_idx_t* topk_idx,
                                                    int* atomic_counter_per_expert,
                                                    int* atomic_finish_counter_per_expert,
                                                    int* next_clean,
                                                    int num_next_clean_int,
                                                    int num_tokens,
                                                    int num_max_dispatch_tokens_per_rank,
                                                    int num_topk,
                                                    int num_experts,
                                                    int rank,
                                                    int num_ranks,
                                                    int num_warp_groups,
                                                    int num_warps_per_group,
                                                    bool round_scale,
                                                    int phases) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto warp_id = thread_id / 32, lane_id = get_lane_id();
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    // May extract UE8M0 from the scales
    using scale_t = std::conditional_t<kUseUE8M0, uint8_t, float>;
    using packed_t = std::conditional_t<kUseUE8M0, uint32_t, float>;
    EP_STATIC_ASSERT(sizeof(packed_t) % sizeof(scale_t) == 0, "Invalid vector length");

    // FP8 staffs
    constexpr int kNumPerChannels = 128;
    const int num_scales = kHidden / kNumPerChannels;
    const size_t hidden_bytes = kHidden * (kUseFP8 ? sizeof(__hip_fp8_storage_t) : sizeof(hip_bfloat16));
    const size_t hidden_int4 = hidden_bytes / sizeof(int4);

    // Message package: index at source (int), 3 reserved int fields, hidden data, FP8 scales
    // NOTES: currently we have 3 reserved int fields for future use
    using vec_t = std::conditional_t<kUseFP8, int2, int4>;
    const size_t num_bytes_per_msg = sizeof(int4) + (kUseFP8 ? (kHidden + num_scales * sizeof(float)) : (kHidden * sizeof(hip_bfloat16)));
    const size_t num_int4_per_msg = num_bytes_per_msg / sizeof(int4);
    EP_DEVICE_ASSERT(num_bytes_per_msg % sizeof(int4) == 0);

    // Expert counts
    constexpr int kNumMaxWarpGroups = 32;
    __shared__ int shared_num_tokens_sent_per_expert[kNumMaxWarpGroups];

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_DISPATCH_RECV;

    // There are 2 kinds of warps in this part:
    // 1. The first-kind warps for FP8 cast and sending top-k tokens
    // 2. The last warp for reading `topk_idx` and count for per-expert information
    if (warp_id < num_warps - 1) {
        constexpr int kNumElemsPerRead = sizeof(int4) / sizeof(hip_bfloat16);
        EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerRead) == 0, "Invalid hidden");
        EP_STATIC_ASSERT(kNumElemsPerRead * 32 % kNumPerChannels == 0, "Invalid vectorization");
        const auto num_threads = (num_warps - 1) * 32;
        const size_t hidden_bf16_int4 = kHidden / kNumElemsPerRead;

        for (int token_idx = sm_id; token_idx < num_tokens; token_idx += num_sms) {
            const auto x_int4 = static_cast<const int4*>(x) + token_idx * hidden_bf16_int4;
            const auto rdma_x_src_idx = reinterpret_cast<int*>(static_cast<uint8_t*>(rdma_x) + token_idx * num_bytes_per_msg);
            const auto rdma_x_vec = reinterpret_cast<vec_t*>(reinterpret_cast<uint8_t*>(rdma_x_src_idx) + sizeof(int4));
            const auto rdma_x_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(rdma_x_vec) + hidden_bytes);

            // Overlap top-k index read and source token index writes
            auto dst_expert_idx = warp_id < num_topk ? static_cast<int>(__ldg(topk_idx + token_idx * num_topk + warp_id)) : -1;
            thread_id == 0 ? (*rdma_x_src_idx = token_idx) : 0;

            // FP8 cast
            EP_STATIC_ASSERT(hidden_bf16_int4 % 32 == 0, "Must use the full warp to reduce");
            #pragma unroll
            for (int i = thread_id; i < hidden_bf16_int4; i += num_threads) {
                // Read
                auto int4_value = __ldg(x_int4 + i);

                if constexpr (kUseFP8) {
                    // Calculate local amax
                    auto bf16_values = reinterpret_cast<hip_bfloat16*>(&int4_value);
                    float fp32_values[kNumElemsPerRead];
                    float amax = kFP8Margin, scale, scale_inv;
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; ++j) {
                        fp32_values[j] = static_cast<float>(bf16_values[j]);
                        amax = fmaxf(amax, fabsf(fp32_values[j]));
                    }

                    // Reduce amax and scale
                    EP_STATIC_ASSERT(kNumElemsPerRead * 32 / kNumPerChannels == 2, "Invalid vectorization");
                    amax = warp_reduce_max<16>(amax);
                    calculate_fp8_scales(amax, scale, scale_inv, round_scale);
                    if (lane_id == 0 or lane_id == 16)
                        rdma_x_scales[i * kNumElemsPerRead / 128] = scale_inv;

                    // Cast into send buffer
                    vec_t int2_value;
                    auto fp8x2_values = reinterpret_cast<__hip_fp8x2_storage_t*>(&int2_value);
                    #pragma unroll
                    for (int j = 0; j < kNumElemsPerRead; j += 2) {
                        float2 fp32x2 = {fp32_values[j] * scale, fp32_values[j + 1] * scale};
                        fp8x2_values[j / 2] = __hip_cvt_float2_to_fp8x2(fp32x2, __HIP_SATFINITE, __HIP_E4M3_FNUZ);
                    }
                    rdma_x_vec[i] = int2_value;
                } else {
                    // Reinterpret-cast is for C++14 compatibility
                    rdma_x_vec[i] = *reinterpret_cast<vec_t*>(&int4_value);
                }
            }
            asm volatile("bar.sync 1, %0;" ::"r"(num_threads));

            // Issue IBGDA sends
            if (dst_expert_idx >= 0) {
                int slot_idx = lane_id == 0 ? atomicAdd(atomic_counter_per_expert + dst_expert_idx, 1) : 0;
                slot_idx = __shfl_sync(0xffffffff, slot_idx, 0);
                const auto dst_rank = dst_expert_idx / num_local_experts;
                const auto dst_expert_local_idx = dst_expert_idx % num_local_experts;
                const auto src_ptr = reinterpret_cast<uint64_t>(rdma_x_src_idx);
                const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                    dst_expert_local_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
                    rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg + slot_idx * num_bytes_per_msg;
                const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                if (not is_rank_masked<true>(mask_buffer_ptr, dst_rank)) {
                    if (dst_p2p_ptr == 0) {
                        nvshmemi_ibgda_put_nbi_warp(dst_ptr, src_ptr, num_bytes_per_msg, dst_rank, dst_expert_local_idx, lane_id, slot_idx);
                    } else {
                        // NOTES: only 2 load iterations for 7K hidden with 8 unrolls
                        const auto* src_int4_ptr = reinterpret_cast<const int4*>(src_ptr);
                        const auto* dst_int4_ptr = reinterpret_cast<int4*>(dst_p2p_ptr);
                        UNROLLED_WARP_COPY(8, lane_id, num_int4_per_msg, dst_int4_ptr, src_int4_ptr, ld_nc_global, st_na_global);
                    }
                }

                // Increase counter after finishing
                __syncwarp();
                lane_id == 0 ? atomic_add_release_global(atomic_finish_counter_per_expert + dst_expert_idx, 1) : 0;
            }
        }
    } else if (warp_id == num_warps - 1) {
        EP_DEVICE_ASSERT(num_sms > 1);
        if (sm_id == 0) {
            // The first SM is also responsible for checking QPs
            // EP_DEVICE_ASSERT(ibgda_get_state()->num_rc_per_pe >= num_local_experts);

            // The first SM is also responsible for cleaning the next buffer
            #pragma unroll
            for (int i = lane_id; i < num_next_clean_int; i += 32)
                next_clean[i] = 0;

            // Notify before executing `int_p`
            __syncwarp();
            #pragma unroll
            for (int i = lane_id; i < num_experts; i += 32)
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG);
        }

        // This SM should be responsible for some destination experts, read `topk_idx` for them
        int expert_count[kNumMaxWarpGroups] = {0};
        const auto expert_begin_idx = sm_id * num_warp_groups;
        const auto expert_end_idx = min(expert_begin_idx + num_warp_groups, num_experts);

        // Per lane count
        #pragma unroll 8
        for (int i = lane_id; i < num_tokens * num_topk; i += 32) {
            auto idx = static_cast<int>(__ldg(topk_idx + i));
            if (idx >= expert_begin_idx and idx < expert_end_idx)
                expert_count[idx - expert_begin_idx]++;
        }

        // Warp reduce
        #pragma unroll
        for (int i = expert_begin_idx; i < expert_end_idx; ++i) {
            auto sum = warp_reduce_sum(expert_count[i - expert_begin_idx]);
            if (lane_id == 0) {
                shared_num_tokens_sent_per_expert[i - expert_begin_idx] = sum;
                atomic_add_release_global(atomic_finish_counter_per_expert + i, FINISHED_SUM_TAG - sum);
            }
        }
    }
    __syncthreads();

    // Issue count sends
    if (responsible_expert_idx < num_experts and sub_warp_id == 0 and lane_id == 0) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto dst_expert_local_idx = responsible_expert_idx % num_local_experts;
        const auto num_tokens_sent = shared_num_tokens_sent_per_expert[responsible_expert_idx - sm_id * num_warp_groups];

        // Wait local sends issued and send expert counts
        while (ld_acquire_global(atomic_finish_counter_per_expert + responsible_expert_idx) != FINISHED_SUM_TAG * 2)
            ;
        auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_count + dst_expert_local_idx * num_ranks + rank);
        auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
        if (not is_rank_masked(mask_buffer_ptr, dst_rank)) {
            if (dst_p2p_ptr == 0) {
                nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), -num_tokens_sent - 1, dst_rank, dst_expert_local_idx);
            } else {
                st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), -num_tokens_sent - 1);
            }
        }

        // Clean workspace for next use
        atomic_counter_per_expert[responsible_expert_idx] = 0;
        atomic_finish_counter_per_expert[responsible_expert_idx] = 0;

        // Clean `packed_recv_count`
        if (dst_rank == 0)
            packed_recv_count[dst_expert_local_idx] = 0;
    }
    __syncwarp();

// Receiving phase
LOW_LATENCY_DISPATCH_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // For send-and-recv kernels, we need a grid sync for making `packed_recv_count` visible
    if (phases & LOW_LATENCY_SEND_PHASE)
        cg::this_grid().sync();

    // Receiving and packing
    if (responsible_expert_idx < num_experts) {
        const auto src_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto rdma_recv_x_uint8 = static_cast<uint8_t*>(rdma_recv_x) +
            local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_msg +
            src_rank * num_max_dispatch_tokens_per_rank * num_bytes_per_msg;
        const auto recv_x_int4 =
            static_cast<int4*>(packed_recv_x) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_int4;
        const auto recv_src_info = packed_recv_src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto recv_range = packed_recv_layout_range + local_expert_idx * num_ranks;
        const auto num_aligned_scales = align_up<int>(num_scales, sizeof(float) / sizeof(scale_t));
        const auto recv_x_scales = static_cast<scale_t*>(packed_recv_x_scales) +
            local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_aligned_scales;

        // Shared between sub-warps in warp groups
        __shared__ int shared_num_recv_tokens[kNumMaxWarpGroups], shared_recv_token_begin_idx[kNumMaxWarpGroups];

        // Wait tokens to arrive
        // NOTES: using sub-warp 1 to overlap with sub-warp 0
        int num_recv_tokens = 0, recv_token_begin_idx;
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 15);
        if (sub_warp_id == 1 and lane_id == 0) {
            auto start_time = clock64();
            uint64_t wait_recv_cost = 0;
            if (not is_rank_masked(mask_buffer_ptr, src_rank)) {
                while ((num_recv_tokens = ld_acquire_sys_global(rdma_recv_count + local_expert_idx * num_ranks + src_rank)) ==
                           0                                                               // data not arrived
                       && (wait_recv_cost = clock64() - start_time) <= NUM_TIMEOUT_CYCLES  // not timeout
                )
                    ;
            }
            // Do not receive tokens if rank timeout or masked
            if (num_recv_tokens == 0)
                num_recv_tokens = -1;
            // Mask rank if timeout
            if (wait_recv_cost > NUM_TIMEOUT_CYCLES) {
                printf("Warning: DeepEP timeout for dispatch receive, rank %d, local_expert_idx %d, src_rank %d\n",
                       rank,
                       local_expert_idx,
                       src_rank);
                if (mask_buffer_ptr == nullptr)
                    trap();
                atomicExch(mask_buffer_ptr + src_rank, 1);
            }

            num_recv_tokens = -num_recv_tokens - 1;
            recv_token_begin_idx = atomicAdd(packed_recv_count + local_expert_idx, num_recv_tokens);
            shared_num_recv_tokens[warp_group_id] = num_recv_tokens;
            shared_recv_token_begin_idx[warp_group_id] = recv_token_begin_idx;
            recv_range[src_rank] = pack2<int, int64_t>(num_recv_tokens, recv_token_begin_idx);

            // Add stats for diagnosis
            if (cumulative_local_expert_recv_stats != nullptr)
                atomicAdd(cumulative_local_expert_recv_stats + local_expert_idx, num_recv_tokens);
            if (dispatch_wait_recv_cost_stats != nullptr)
                atomicAdd(reinterpret_cast<unsigned long long*>(dispatch_wait_recv_cost_stats + src_rank), wait_recv_cost);
        }
        asm volatile("bar.sync %0, %1;" ::"r"(warp_group_id + 2), "r"(num_warps_per_group * 32));
        num_recv_tokens = shared_num_recv_tokens[warp_group_id];
        recv_token_begin_idx = shared_recv_token_begin_idx[warp_group_id];

        // Copy tokens
        EP_DEVICE_ASSERT(num_scales <= 64);
        for (int i = sub_warp_id; i < num_recv_tokens; i += num_warps_per_group) {
            // Copy source info
            const auto src_src_idx = reinterpret_cast<int*>(rdma_recv_x_uint8 + i * num_bytes_per_msg);
            if (lane_id == 0)
                recv_src_info[recv_token_begin_idx + i] = ld_nc_global(src_src_idx);
            __syncwarp();

            // Copy data
            // NOTES: only 2 load iterations for 7K hidden with 7 unrolls
            const auto src_data = reinterpret_cast<int4*>(reinterpret_cast<uint8_t*>(src_src_idx) + sizeof(int4));
            const auto dst_data = recv_x_int4 + (recv_token_begin_idx + i) * hidden_int4;
            UNROLLED_WARP_COPY(7, lane_id, hidden_int4, dst_data, src_data, ld_nc_global, st_na_global);

            // Copy scales
            if constexpr (kUseFP8) {
                // Equivalent CuTe layout:
                //   (num_tokens, (num_packed, num_elems_per_pack)):(num_elems_per_pack, (num_tokens * num_elems_per_pack, 1))
                const auto src_scales = reinterpret_cast<float*>(reinterpret_cast<uint8_t*>(src_data) + hidden_bytes);
                const auto num_elems_per_pack = static_cast<int>(sizeof(packed_t) / sizeof(scale_t));
                const auto token_idx = recv_token_begin_idx + i;
                const auto token_stride = num_elems_per_pack;
                const auto pack_stride = num_ranks * num_max_dispatch_tokens_per_rank * num_elems_per_pack;
                if (lane_id < num_scales) {
                    const auto pack_idx = lane_id / num_elems_per_pack;
                    const auto elem_idx = lane_id % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(ld_nc_global(src_scales + lane_id));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
                if (lane_id + 32 < num_scales) {
                    const auto pack_idx = (lane_id + 32) / num_elems_per_pack;
                    const auto elem_idx = (lane_id + 32) % num_elems_per_pack;
                    auto scale = extract_required_scale_format<kUseUE8M0>(ld_nc_global(src_scales + lane_id + 32));
                    recv_x_scales[token_idx * token_stride + pack_idx * pack_stride + elem_idx] = scale;
                }
            }
        }
    }
}

void dispatch(void* packed_recv_x,
              void* packed_recv_x_scales,
              int* packed_recv_src_info,
              int64_t* packed_recv_layout_range,
              int* packed_recv_count,
              int* mask_buffer_ptr,
              int* cumulative_local_expert_recv_stats,
              int64_t* dispatch_wait_recv_cost_stats,
              void* rdma_recv_x,
              int* rdma_recv_count,
              void* rdma_x,
              const void* x,
              const topk_idx_t* topk_idx,
              int* next_clean,
              int num_next_clean_int,
              int num_tokens,
              int hidden,
              int num_max_dispatch_tokens_per_rank,
              int num_topk,
              int num_experts,
              int rank,
              int num_ranks,
              bool use_fp8,
              bool round_scale,
              bool use_ue8m0,
              void* workspace,
              int num_device_sms,
              hipStream_t stream,
              int phases) {
    constexpr int kNumMaxTopK = 11;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0);
    EP_HOST_ASSERT(kNumMaxTopK + 1 <= num_warp_groups * num_warps_per_group);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms = ceil_div(num_experts, num_warp_groups);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopK);

    // Workspace checks
    auto atomic_counter_per_expert = static_cast<int*>(workspace);
    auto atomic_finish_counter_per_expert = atomic_counter_per_expert + num_experts;
    EP_HOST_ASSERT(num_experts * sizeof(int) * 2 <= NUM_WORKSPACE_BYTES);

    // FP8 checks
    if (use_ue8m0)
        EP_HOST_ASSERT(round_scale and "UE8M0 SF requires `round_scale=True`");

#define DISPATCH_LAUNCH_CASE(hidden)                         \
    {                                                        \
        auto dispatch_func = dispatch<false, false, hidden>; \
        if (use_fp8 and not use_ue8m0)                       \
            dispatch_func = dispatch<true, false, hidden>;   \
        if (use_fp8 and use_ue8m0)                           \
            dispatch_func = dispatch<true, true, hidden>;    \
        LAUNCH_KERNEL(&cfg,                                  \
                      dispatch_func,                         \
                      packed_recv_x,                         \
                      packed_recv_x_scales,                  \
                      packed_recv_src_info,                  \
                      packed_recv_layout_range,              \
                      packed_recv_count,                     \
                      mask_buffer_ptr,                       \
                      cumulative_local_expert_recv_stats,    \
                      dispatch_wait_recv_cost_stats,         \
                      rdma_recv_x,                           \
                      rdma_recv_count,                       \
                      rdma_x,                                \
                      x,                                     \
                      topk_idx,                              \
                      atomic_counter_per_expert,             \
                      atomic_finish_counter_per_expert,      \
                      next_clean,                            \
                      num_next_clean_int,                    \
                      num_tokens,                            \
                      num_max_dispatch_tokens_per_rank,      \
                      num_topk,                              \
                      num_experts,                           \
                      rank,                                  \
                      num_ranks,                             \
                      num_warp_groups,                       \
                      num_warps_per_group,                   \
                      round_scale,                           \
                      phases);                               \
    }                                                        \
    break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
    SWITCH_HIDDEN(DISPATCH_LAUNCH_CASE);
#undef DISPATCH_LAUNCH_CASE
}

template <int kNumSendUnrolls>
__forceinline__ __device__ int logfmt_encode(void* buffer, nv_bfloat162* shared_amaxmin, const int& lane_id) {
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32;  // `== log_2(2 ^ (2 ^ 5))`
    constexpr int kNumBits = 10;
    constexpr int kNumValues = 1 << (kNumBits - 1);

    int4 int4_values[kNumSendUnrolls];
    const auto& uint32_values = reinterpret_cast<uint32_t*>(int4_values);
    const auto& bf162_values = reinterpret_cast<nv_bfloat162*>(int4_values);

    // Calculate lane offset
    const auto& ld_buffer = reinterpret_cast<uint32_t*>(static_cast<uint8_t*>(buffer) + lane_id * (kNumSendUnrolls * sizeof(int4)));
    const auto& st_buffer =
        reinterpret_cast<uint32_t*>(static_cast<uint8_t*>(buffer) + lane_id * (kNumSendUnrolls * sizeof(int4) * 10 / 16));

    // Local log amax
    auto bf162_amax = __hip_bfloat162(CUDART_ZERO_BF16, CUDART_ZERO_BF16);
    auto bf162_amin = __hip_bfloat162(CUDART_INF_BF16, CUDART_INF_BF16);
    uint32_t local_signs = 0;
    #pragma unroll
    for (int k = 0; k < kNumSendUnrolls * kNumElemsPerInt4 / 2; ++k) {
        // TODO: eliminate bank conflicts
        uint32_values[k] = ld_buffer[k];
        local_signs |= ((uint32_values[k] >> 15) & 1) << (k * 2);
        local_signs |= ((uint32_values[k] >> 31) & 1) << (k * 2 + 1);
        uint32_values[k] &= 0x7fff7fff;

        bf162_amax = __hmax2(bf162_amax, bf162_values[k]);
        bf162_amin = __hmin2(bf162_amin, bf162_values[k]);
    }

    // Reduce per 128 channels
    // TODO: figure out how hardware do 2-byte min/max
    auto amax = std::max(static_cast<float>(bf162_amax.x), static_cast<float>(bf162_amax.y));
    auto amin = std::min(static_cast<float>(bf162_amin.x), static_cast<float>(bf162_amin.y));
    constexpr static int kNumLanesToReduce = 128 * sizeof(hip_bfloat16) / (kNumSendUnrolls * sizeof(int4));
    amax = warp_reduce_max<kNumLanesToReduce>(amax);
    amin = warp_reduce_min<kNumLanesToReduce>(amin);

    // Write min/max into the shared memory
    if (shared_amaxmin != nullptr)
        *shared_amaxmin = __hip_bfloat162(amax, amin);
    __syncwarp();

    // Calculate log amin/amax float
    const auto& log_amax = log2f_approx(amax);
    const auto& log_amin = fmaxf(log2f_approx(amin), log_amax - kMinClip);
    const bool& enable_cast = warp_reduce_and<kNumLanesToReduce, true>(log_amax < kLogThreshold and log_amin < log_amax);

    // Case into LogFMT-10 if satisfied
    if (enable_cast) {
        const auto step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        const auto step_inv = 1.0f / step;
        const auto rounding = 2.0f - log2f_approx((1.0f + exp2f_approx(step)) * 0.5f) * step_inv;
        const auto fused_rounding = rounding - log_amin * step_inv;

        // Pack every 256 bits into 160 bits
        EP_STATIC_ASSERT(kNumSendUnrolls == 2 or kNumSendUnrolls == 4, "kNumSendUnrolls == 2 or 4 only");
        uint32_t encoded[kNumElemsPerInt4 * 2];
        #pragma unroll 1
        for (int i = 0; i < kNumSendUnrolls / 2; ++i) {
            #pragma unroll
            for (int k = 0; k < kNumElemsPerInt4; ++k) {
                const auto& [x, y] = __bfloat1622float2(bf162_values[i * kNumElemsPerInt4 + k]);
                encoded[k * 2 + 0] = __float2uint_rd(fmaxf(log2f_approx(x) * step_inv + fused_rounding, 0));
                encoded[k * 2 + 1] = __float2uint_rd(fmaxf(log2f_approx(y) * step_inv + fused_rounding, 0));
            }
            st_buffer[i * 5 + 0] = (encoded[0] >> 0) | (encoded[1] << 9) | (encoded[2] << 18) | (encoded[3] << 27);
            st_buffer[i * 5 + 1] = (encoded[3] >> 5) | (encoded[4] << 4) | (encoded[5] << 13) | (encoded[6] << 22) | (encoded[7] << 31);
            st_buffer[i * 5 + 2] = (encoded[7] >> 1) | (encoded[8] << 8) | (encoded[9] << 17) | (encoded[10] << 26);
            st_buffer[i * 5 + 3] =
                (encoded[10] >> 6) | (encoded[11] << 3) | (encoded[12] << 12) | (encoded[13] << 21) | (encoded[14] << 30);
            st_buffer[i * 5 + 4] = (encoded[14] >> 2) | (encoded[15] << 7) | ((i == 0) ? (local_signs << 16) : (local_signs & 0xffff0000u));
        }
        tma_store_fence();
        __syncwarp();
    }

    // Return TMA copy bytes
    return enable_cast ? (32 * (kNumSendUnrolls * sizeof(int4) * 8 * 10 / 16 / 8)) : (32 * (kNumSendUnrolls * sizeof(int4)));
}

template <int kNumLanes, int kNumSendUnrolls, int kNumRecvUnrolls>
__forceinline__ __device__ void logfmt_check_amaxmin(
    uint8_t* meta_buffer, float2* shared_log_amax, float2* shared_log_amin, int* shared_cast_info, const int lane_id) {
    constexpr float kLogThreshold = 0;
    constexpr float kMinClip = 32;  // `== log_2(2 ^ (2 ^ 5))`

    bool enable_cast = true;
    if (lane_id < kNumLanes) {
        // Calculate log amin/amax float
        auto amaxmin2 = reinterpret_cast<uint64_t*>(meta_buffer)[lane_id];
        const auto& bf162_amaxmin = reinterpret_cast<__hip_bfloat162*>(&amaxmin2);
        float log_amax[2], log_amin[2];
        #pragma unroll
        for (int i = 0; i < 2; ++i) {
            auto amax = static_cast<float>(bf162_amaxmin[i].x);
            auto amin = static_cast<float>(bf162_amaxmin[i].y);
            log_amax[i] = log2f_approx(amax);
            log_amin[i] = amin == 0 ? log_amax[i] - kMinClip : fmaxf(log2f_approx(amin), log_amax[i] - kMinClip);
            enable_cast = enable_cast and log_amax[i] < kLogThreshold and log_amin[i] < log_amax[i];
        }
        shared_log_amax[lane_id] = make_float2(log_amax[0], log_amax[1]);
        shared_log_amin[lane_id] = make_float2(log_amin[0], log_amin[1]);
    }

    const auto& casted = warp_reduce_and<kNumSendUnrolls>(enable_cast) ? 1u << (lane_id / kNumRecvUnrolls) : 0u;
    const auto& num_casted_prefix = __popc(warp_reduce_or<kNumRecvUnrolls, true>(casted) & ((1u << (lane_id / kNumRecvUnrolls)) - 1));

    if (lane_id < kNumLanes and lane_id % kNumRecvUnrolls == 0)
        shared_cast_info[lane_id / kNumRecvUnrolls] = (num_casted_prefix << 1) | (casted ? 1u : 0u);
    __syncwarp();
}

template <int kNumRecvUnrolls>
__forceinline__ __device__ void decode_and_accumulate(
    uint32_t* ld_buffer, float* accum, const float& log_amax, const float& log_amin, const bool& enable_cast, const float& weight) {
    if (enable_cast) {
        constexpr int kNumBits = 10;
        constexpr int kNumValues = 1 << (kNumBits - 1);

        const auto& step = (log_amax - log_amin) / static_cast<float>(kNumValues - 2);
        auto decode = [=](const uint32_t& encoded, const uint32_t& sign) {
            const auto decoded = encoded == 0 ? .0f : exp2f_approx((encoded - 1) * step + log_amin);
            return sign ? -decoded : decoded;
        };

        EP_STATIC_ASSERT(kNumRecvUnrolls == 2 or kNumRecvUnrolls == 4, "kNumRecvUnrolls == 2 or 4 only");
        #pragma unroll
        for (int i = 0; i < kNumRecvUnrolls / 2; ++i) {
            uint32_t concat[6];
            concat[0] = ld_buffer[i * 5];
            #pragma unroll
            for (int k = 1; k < 5; ++k)
                concat[k] = (ld_buffer[i * 5 + k - 1] >> (32 - k * 5)) | (ld_buffer[i * 5 + k] << (k * 5));
            concat[5] = ld_buffer[i * 5 + 4] >> 7;

            const uint32_t& local_signs = ld_buffer[i * 5 + 4] >> 16;
            #pragma unroll
            for (int k = 0; k < 5; ++k) {
                accum[i * 16 + k * 3 + 0] += decode((concat[k] >> 0) & 0x1ff, (local_signs >> (k * 3 + 0)) & 1) * weight;
                accum[i * 16 + k * 3 + 1] += decode((concat[k] >> 9) & 0x1ff, (local_signs >> (k * 3 + 1)) & 1) * weight;
                accum[i * 16 + k * 3 + 2] += decode((concat[k] >> 18) & 0x1ff, (local_signs >> (k * 3 + 2)) & 1) * weight;
            }
            accum[i * 16 + 15] += decode(concat[5] & 0x1ff, (local_signs >> 15) & 1) * weight;
        }
    } else {
        #pragma unroll
        for (int k = 0; k < kNumRecvUnrolls * 4; ++k) {
            auto bf16_pack = *reinterpret_cast<__hip_bfloat162*>(ld_buffer + k);
            accum[k * 2 + 0] += static_cast<float>(bf16_pack.x) * weight;
            accum[k * 2 + 1] += static_cast<float>(bf16_pack.y) * weight;
        }
    }
}

template <bool kUseLogFMT, int kHidden, int kNumMaxTopk, int kNumMaxUnrolls>
__global__ __launch_bounds__(1024, 1) void combine(void* combined_x,
                                                   void* rdma_recv_x,
                                                   int* rdma_recv_flag,
                                                   void* rdma_send_x,
                                                   const void* x,
                                                   const topk_idx_t* topk_idx,
                                                   const float* topk_weights,
                                                   const int* src_info,
                                                   const int64_t* layout_range,
                                                   int* mask_buffer_ptr,
                                                   int64_t* combine_wait_recv_cost_stats,
                                                   int* next_clean,
                                                   int num_next_clean_int,
                                                   int* atomic_clean_flag,
                                                   int num_combined_tokens,
                                                   int hidden,
                                                   int num_topk,
                                                   int num_max_dispatch_tokens_per_rank,
                                                   int num_experts,
                                                   int rank,
                                                   int num_ranks,
                                                   int num_warp_groups,
                                                   int num_warps_per_group,
                                                   int phases,
                                                   bool zero_copy) {
    const auto sm_id = __shfl_sync(0xffffffff, static_cast<int>(blockIdx.x), 0);
    const auto num_sms = __shfl_sync(0xffffffff, static_cast<int>(gridDim.x), 0);
    const auto thread_id = static_cast<int>(threadIdx.x);
    const auto num_threads = __shfl_sync(0xffffffff, static_cast<int>(blockDim.x), 0);
    const auto warp_id = __shfl_sync(0xffffffff, thread_id / 32, 0), lane_id = get_lane_id();
    const auto num_local_experts = num_experts / num_ranks;
    const auto warp_group_id = warp_id / num_warps_per_group;
    const auto sub_warp_id = warp_id % num_warps_per_group;
    const auto responsible_expert_idx = sm_id * num_warp_groups + warp_group_id;

    extern __shared__ __align__(1024) uint8_t smem_buffer[];

    // Data type staffs
    constexpr int kNumElemsPerInt4 = sizeof(int4) / sizeof(hip_bfloat16);
    constexpr int64_t hidden_bf16_int4 = kHidden / kNumElemsPerInt4;

    // Use different unroll factors for send and recv phases
    constexpr int kNumSendUnrolls = kHidden % (32 * 4 * sizeof(int4) / sizeof(hip_bfloat16)) == 0 ? 4 : 2;
    constexpr int kNumRecvUnrolls = 2;
    constexpr int hidden_bf16_int4_pad = align_up(static_cast<int>(hidden_bf16_int4), 32 * kNumSendUnrolls);
    EP_STATIC_ASSERT(kHidden % (32 * 2 * sizeof(int4) / sizeof(hip_bfloat16)) == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumSendUnrolls <= kNumMaxUnrolls and kNumRecvUnrolls <= kNumMaxUnrolls, "Invalid unrolls");
    EP_STATIC_ASSERT(hidden_bf16_int4 % kNumSendUnrolls == 0, "Invalid hidden");
    EP_STATIC_ASSERT(kNumSendUnrolls >= kNumRecvUnrolls, "Invalid unroll factors");

    // Message package
    EP_STATIC_ASSERT(kHidden % 128 == 0, "Invalid hidden");
    constexpr int kNumDivisions = kHidden / 128;
    constexpr int kNumMetaBytes = kNumDivisions * sizeof(nv_bfloat162);
    constexpr size_t num_bytes_per_slot = kHidden * sizeof(hip_bfloat16) + kNumMetaBytes;
    EP_STATIC_ASSERT(num_bytes_per_slot % sizeof(int4) == 0, "Invalid vectorization");

    // Sending phase
    if ((phases & LOW_LATENCY_SEND_PHASE) == 0)
        goto LOW_LATENCY_COMBINE_RECV;

    // Clean up next buffer
    if (sm_id == 0 and warp_group_id == 0 and sub_warp_id == 0) {
        #pragma unroll
        for (int i = lane_id; i < num_next_clean_int; i += 32)
            next_clean[i] = 0;

        // Notify before executing `int_p`
        __syncwarp();
        if (lane_id == 0)
            atomic_add_release_global(atomic_clean_flag, num_experts);
    }

    // Issue IBGDA sends
    if (responsible_expert_idx < num_experts) {
        const auto dst_rank = responsible_expert_idx / num_local_experts;
        const auto local_expert_idx = responsible_expert_idx % num_local_experts;
        const auto global_expert_idx = rank * num_local_experts + local_expert_idx;
        const auto layout = __ldg(layout_range + local_expert_idx * num_ranks + dst_rank);
        const auto local_x =
            static_cast<const int4*>(x) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * hidden_bf16_int4;
        const auto local_src_info = src_info + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank;
        const auto rdma_send_x_vec =
            static_cast<uint8_t*>(rdma_send_x) + local_expert_idx * num_ranks * num_max_dispatch_tokens_per_rank * num_bytes_per_slot;

        // Unpack layout
        int offset, num_tokens_to_send;
        unpack2(layout, num_tokens_to_send, offset);

        // TMA stuffs
        constexpr int kNumTMABufferBytes = sizeof(int4) * 32 * kNumSendUnrolls;
        constexpr int kNumStages = 3;
        constexpr int kNumPrefetch = 1;
        EP_STATIC_ASSERT(kNumStages == 3 and kNumPrefetch == 1, "Invalid stages");

        auto smem_ptr = smem_buffer + warp_id * (kNumStages * (kNumTMABufferBytes + 16) + kNumMetaBytes);
        uint32_t tma_phase = 0;
        auto tma_buffers = PatternVisitor([=](const int& i) { return reinterpret_cast<int4*>(smem_ptr + i * (kNumTMABufferBytes + 16)); });
        auto full_barriers = PatternVisitor(
            [=](const int& i) { return reinterpret_cast<uint64_t*>(smem_ptr + i * (kNumTMABufferBytes + 16) + kNumTMABufferBytes); });
        auto meta_buffers = kUseLogFMT ? reinterpret_cast<nv_bfloat162*>(smem_ptr + kNumStages * (kNumTMABufferBytes + 16)) : nullptr;
        EP_STATIC_ASSERT(kNumSendUnrolls * kNumStages <= 12, "TMA buffer size exceed limit");

        // Initialize m-barriers
        if (lane_id < kNumStages) {
            mbarrier_init(full_barriers[lane_id], 1);
            fence_barrier_init();
        }
        __syncwarp();

        constexpr int kNumIters = hidden_bf16_int4_pad / (32 * kNumSendUnrolls);
        auto tma_load_and_arrive = [&](const int& stage_idx, const int4* gmem_ptr, const int& num_bytes) {
            tma_load_1d(tma_buffers[stage_idx], gmem_ptr, full_barriers[stage_idx], num_bytes);
            mbarrier_arrive_and_expect_tx(full_barriers[stage_idx], num_bytes);
        };
        auto get_num_tma_bytes = [&](const int& offset_int4) {
            return min(kNumTMABufferBytes, static_cast<int>((hidden_bf16_int4 - offset_int4) * sizeof(int4)));
        };

        // Issue IBGDA send
        if (not is_rank_masked<true>(mask_buffer_ptr, dst_rank)) {
            for (int token_idx = offset + sub_warp_id; token_idx < offset + num_tokens_to_send; token_idx += num_warps_per_group) {
                const auto x_int4 = local_x + token_idx * hidden_bf16_int4;
                const auto rdma_send_type_row = reinterpret_cast<int*>(rdma_send_x_vec + token_idx * num_bytes_per_slot);
                const auto rdma_send_x_vec_row = reinterpret_cast<uint8_t*>(rdma_send_type_row);

                // Copy directly to local rank, or copy to buffer and issue RDMA
                const auto src_idx = __shfl_sync(0xffffffff, __ldg(local_src_info + token_idx), 0);
                const auto buf_ptr = reinterpret_cast<int64_t>(rdma_send_x_vec_row);
                const auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_x) +
                    (global_expert_idx * num_max_dispatch_tokens_per_rank + src_idx) * num_bytes_per_slot;
                const auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
                int num_send_bytes = hidden * sizeof(hip_bfloat16);

                if (not zero_copy or dst_p2p_ptr != 0) {
                    // Read from `cpy_src_int4_ptr` and copy into `cpy_dst_int4_ptr`
                    const auto cpy_src_int4_ptr = zero_copy ? reinterpret_cast<int4*>(buf_ptr) : x_int4;
                    const auto cpy_dst_int4_ptr =
                        dst_p2p_ptr == 0 ? reinterpret_cast<int4*>(buf_ptr) : reinterpret_cast<int4*>(dst_p2p_ptr);

                    // Prefetch
                    if (elect_one_sync())
                        tma_load_and_arrive(0, cpy_src_int4_ptr, get_num_tma_bytes(0));
                    __syncwarp();

                    int tma_offset_bytes = kNumMetaBytes;
                    #pragma unroll
                    for (int i = lane_id * kNumSendUnrolls, iter_idx = 0; i < hidden_bf16_int4_pad; i += 32 * kNumSendUnrolls, ++iter_idx) {
                        // Load the next iteration
                        const int& stage_idx = iter_idx % kNumStages;
                        const int& next_stage_idx = (iter_idx + 1) % kNumStages;
                        if (iter_idx + 1 < kNumIters and elect_one_sync()) {
                            tma_store_wait<kNumStages - kNumPrefetch - 1>();
                            const auto& offset_int4 = i + 32 * kNumSendUnrolls;
                            tma_load_and_arrive(next_stage_idx, cpy_src_int4_ptr + offset_int4, get_num_tma_bytes(offset_int4));
                        }
                        __syncwarp();

                        // Wait the current TMA arrival
                        EP_STATIC_ASSERT(kNumStages < 32, "Too many stages");
                        mbarrier_wait<true>(full_barriers[stage_idx], tma_phase, stage_idx);
                        if constexpr (kUseLogFMT) {
                            // Cast if possible
                            constexpr int kNumInt4PerDivision = 128 / kNumElemsPerInt4;
                            int num_tma_bytes = logfmt_encode<kNumSendUnrolls>(
                                tma_buffers[stage_idx],
                                // NOTES: only the leader lane will write the result
                                (i % kNumInt4PerDivision == 0) ? meta_buffers + i / kNumInt4PerDivision : nullptr,
                                lane_id);
                            if (elect_one_sync())
                                tma_store_1d(
                                    tma_buffers[stage_idx], reinterpret_cast<uint8_t*>(cpy_dst_int4_ptr) + tma_offset_bytes, num_tma_bytes);
                            tma_offset_bytes += num_tma_bytes;
                        } else {
                            // BF16 original values
                            if (elect_one_sync())
                                tma_store_1d(tma_buffers[stage_idx], cpy_dst_int4_ptr + i, get_num_tma_bytes(i));
                        }
                        __syncwarp();
                    }

                    // Store metadata (min/max values) for LogFMT
                    if constexpr (kUseLogFMT) {
                        num_send_bytes = tma_offset_bytes;
                        if (elect_one_sync())
                            tma_store_1d(meta_buffers, cpy_dst_int4_ptr, kNumMetaBytes);
                    }

                    // Flush all stores
                    tma_store_wait<0>();
                    __syncwarp();
                }

                // Issue RDMA
                // NOTES: for zero-copy mode, we assume the data is already in the send buffer
                if (dst_p2p_ptr == 0)
                    nvshmemi_ibgda_put_nbi_warp(dst_ptr, buf_ptr, num_send_bytes, dst_rank, local_expert_idx, lane_id, token_idx - offset);
            }
        }

        // Put the finishing flag
        EP_DEVICE_ASSERT(num_warps_per_group > 1 and num_warp_groups < 16);
        asm volatile("bar.sync %0, %1;" ::"r"(warp_group_id + 1), "r"(num_warps_per_group * 32));
        if (sub_warp_id == 1 and lane_id == 0) {
            while (ld_acquire_global(atomic_clean_flag) == 0)
                ;
            auto dst_ptr = reinterpret_cast<uint64_t>(rdma_recv_flag + global_expert_idx);
            auto dst_p2p_ptr = nvshmemi_get_p2p_ptr(dst_ptr, rank, dst_rank);
            if (not is_rank_masked(mask_buffer_ptr, dst_rank)) {
                if (dst_p2p_ptr == 0) {
                    nvshmemi_ibgda_amo_nonfetch_add(reinterpret_cast<int*>(dst_ptr), 1, dst_rank, local_expert_idx);
                } else {
                    st_release_sys_global(reinterpret_cast<int*>(dst_p2p_ptr), 1);
                }
            }
            atomic_add_release_global(atomic_clean_flag, -1);
        }
        __syncwarp();

        // Destroy m-barriers
        if (lane_id < kNumStages) {
            mbarrier_inval(full_barriers[lane_id]);
            fence_barrier_init();
        }
        __syncwarp();
    }

// Receiving phase
LOW_LATENCY_COMBINE_RECV:
    if ((phases & LOW_LATENCY_RECV_PHASE) == 0)
        return;

    // Wait all ranks to arrive
    if (responsible_expert_idx < num_experts) {
        EP_DEVICE_ASSERT(num_warps_per_group > 1);
        if (sub_warp_id == 0 and lane_id == 0) {
            const auto src_rank = responsible_expert_idx / num_local_experts;
            auto start_time = clock64();
            uint64_t wait_recv_cost = 0;
            if (not is_rank_masked(mask_buffer_ptr, src_rank)) {
                while (ld_acquire_sys_global(rdma_recv_flag + responsible_expert_idx) == 0  // recv not ready
                       && (wait_recv_cost = clock64() - start_time) <= NUM_TIMEOUT_CYCLES   // not timeout
                )
                    ;
            }
            // Mask rank if timeout
            if (wait_recv_cost > NUM_TIMEOUT_CYCLES) {
                printf("Warning: DeepEP timeout for combine receive, rank %d, local_expert_idx %d, src_rank %d\n",
                       rank,
                       responsible_expert_idx % num_local_experts,
                       src_rank);
                if (mask_buffer_ptr == nullptr)
                    trap();
                atomicExch(mask_buffer_ptr + src_rank, 1);
            }

            if (combine_wait_recv_cost_stats != nullptr) {
                atomicAdd(reinterpret_cast<unsigned long long*>(combine_wait_recv_cost_stats + src_rank), wait_recv_cost);
            }
        }
    }
    cg::this_grid().sync();

    // Reassign warp groups
    constexpr int kMaxNumGroups = 2;
    const int num_decode_warps = hidden_bf16_int4_pad / (kNumRecvUnrolls * 32);
    const int num_groups = min(kMaxNumGroups, (num_threads / 32) / (num_decode_warps + 1));
    const int decode_warp_idx = __shfl_sync(0xffffffff, warp_id % (num_decode_warps + 1), 0);
    const int group_idx = __shfl_sync(0xffffffff, warp_id / (num_decode_warps + 1), 0);
    EP_STATIC_ASSERT(kHidden % (32 * kNumElemsPerInt4) == 0, "Invalid vectorization");
    EP_DEVICE_ASSERT(num_topk <= 32);
    EP_DEVICE_ASSERT(num_groups > 0);

    if (group_idx < num_groups) {
        constexpr int kNumStages = 3;
        constexpr int kNumTMABufferBytes = 16 * 2 + kHidden * 2;
        constexpr int kNumBF16PerWarpBytes = 32 * kNumRecvUnrolls * kNumElemsPerInt4 * 2;
        constexpr int kNumLogFMTPerWarpBytes = kNumBF16PerWarpBytes / 16 * 10;
        constexpr int kNumDivisionBytes = kNumDivisions * sizeof(uint32_t);
        constexpr int kNumBytesPerGroup = kNumStages * kNumTMABufferBytes + kHidden * 2 + kNumStages * kNumDivisionBytes * 3;

        // Reallocate shared memory
        const auto smem_group_buffer = smem_buffer + kNumBytesPerGroup * group_idx;
        auto full_barriers =
            PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_group_buffer + i * kNumTMABufferBytes); });
        auto empty_barriers =
            PatternVisitor([=](const int& i) { return reinterpret_cast<uint64_t*>(smem_group_buffer + i * kNumTMABufferBytes + 8); });
        auto tma_ld_buffers =
            PatternVisitor([=](const int& i) { return reinterpret_cast<uint8_t*>(smem_group_buffer + i * kNumTMABufferBytes + 16); });
        auto tma_st_buffers = PatternVisitor([=](const int& i) {
            return reinterpret_cast<uint32_t*>(smem_group_buffer + kNumStages * kNumTMABufferBytes + i * kNumBF16PerWarpBytes);
        });

        // Redundant when logfmt is disabled
        const auto smem_group_ptr = smem_group_buffer + kNumStages * kNumTMABufferBytes + kHidden * 2;
        auto log_amax_buffers =
            PatternVisitor([=](const int& i) { return reinterpret_cast<float*>(smem_group_ptr + i * kNumDivisionBytes); });
        auto log_amin_buffers = PatternVisitor([=](const int& i) {
            return reinterpret_cast<float*>(smem_group_ptr + kNumStages * kNumDivisionBytes + i * kNumDivisionBytes);
        });
        auto cast_info_buffers = PatternVisitor([=](const int& i) {
            return reinterpret_cast<int*>(smem_group_ptr + kNumStages * kNumDivisionBytes * 2 + i * kNumDivisionBytes);
        });

        uint32_t tma_phase = 0;
        EP_STATIC_ASSERT(kNumStages < 32, "Too many stages");
        if (decode_warp_idx == num_decode_warps)
            tma_phase = (1 << kNumStages) - 1;

        // Initialize m-barriers
        if (decode_warp_idx == num_decode_warps and lane_id < kNumStages) {
            mbarrier_init(full_barriers[lane_id], 1);
            mbarrier_init(empty_barriers[lane_id], num_decode_warps);
        }
        asm volatile("bar.sync %0, %1;" ::"r"(group_idx + 1), "r"((num_decode_warps + 1) * 32));

        int stage_idx = 0, topk_idx_by_lane = 0;
        EP_STATIC_ASSERT(kNumMaxTopk <= 32, "Invalid number of topks");
        if (decode_warp_idx == num_decode_warps) {
            // TMA load warp
            for (int token_idx = sm_id + num_sms * group_idx; token_idx < num_combined_tokens; token_idx += num_sms * num_groups) {
                if (lane_id < num_topk)
                    topk_idx_by_lane = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + lane_id));
                for (int i = 0; i < num_topk; ++i) {
                    int topk_idx_reg = __shfl_sync(0xffffffff, topk_idx_by_lane, i);
                    if (topk_idx_reg < 0)
                        continue;
                    if (is_rank_masked(mask_buffer_ptr, topk_idx_reg / num_local_experts))
                        continue;

                    mbarrier_wait<true>(empty_barriers[stage_idx], tma_phase, stage_idx);
                    auto buffer = static_cast<uint8_t*>(rdma_recv_x) +
                        (topk_idx_reg * num_max_dispatch_tokens_per_rank + token_idx) * num_bytes_per_slot;
                    if constexpr (kUseLogFMT) {
                        logfmt_check_amaxmin<kNumDivisions / 2, kNumSendUnrolls, kNumRecvUnrolls>(
                            buffer,
                            reinterpret_cast<float2*>(log_amax_buffers[stage_idx]),
                            reinterpret_cast<float2*>(log_amin_buffers[stage_idx]),
                            cast_info_buffers[stage_idx],
                            lane_id);
                    }
                    if (elect_one_sync()) {
                        int num_casted = 0;
                        if constexpr (kUseLogFMT) {
                            const auto& info = cast_info_buffers[stage_idx][num_decode_warps - 1];
                            num_casted = (info >> 1) + (info & 1);
                        }
                        int num_tma_bytes = num_casted * kNumLogFMTPerWarpBytes + (num_decode_warps - num_casted) * kNumBF16PerWarpBytes;
                        tma_load_1d(
                            tma_ld_buffers[stage_idx], buffer + (kUseLogFMT ? kNumMetaBytes : 0), full_barriers[stage_idx], num_tma_bytes);
                        mbarrier_arrive_and_expect_tx(full_barriers[stage_idx], num_tma_bytes);
                    }
                    __syncwarp();
                    stage_idx = (stage_idx + 1) % kNumStages;
                }
            }
        } else {
            // Reduction warps
            float topk_weights_by_lane;
            for (int token_idx = sm_id + num_sms * group_idx; token_idx < num_combined_tokens; token_idx += num_sms * num_groups) {
                if (lane_id < num_topk) {
                    topk_idx_by_lane = static_cast<int>(__ldg(topk_idx + token_idx * num_topk + lane_id));
                    topk_weights_by_lane = __ldg(topk_weights + token_idx * num_topk + lane_id);
                }
                __syncwarp();

                float combined_values[kNumElemsPerInt4 * kNumRecvUnrolls] = {0.0f};
                for (int i = 0; i < num_topk; ++i) {
                    int topk_idx_reg = __shfl_sync(0xffffffff, topk_idx_by_lane, i);
                    if (topk_idx_reg < 0)
                        continue;
                    if (is_rank_masked(mask_buffer_ptr, topk_idx_reg / num_local_experts))
                        continue;
                    const auto& topk_weight = __shfl_sync(0xffffffff, topk_weights_by_lane, i);

                    mbarrier_wait<true>(full_barriers[stage_idx], tma_phase, stage_idx);
                    if constexpr (kUseLogFMT) {
                        const auto& info = cast_info_buffers[stage_idx][decode_warp_idx];
                        bool enable_cast = info & 1;
                        int num_casted_prefix = info >> 1;
                        int tma_offset =
                            kNumLogFMTPerWarpBytes * num_casted_prefix + kNumBF16PerWarpBytes * (decode_warp_idx - num_casted_prefix);
                        int division_idx = decode_warp_idx * (kNumRecvUnrolls * 2) + lane_id * kNumRecvUnrolls / 16;
                        decode_and_accumulate<kNumRecvUnrolls>(
                            reinterpret_cast<uint32_t*>(tma_ld_buffers[stage_idx] + tma_offset +
                                                        (enable_cast ? kNumLogFMTPerWarpBytes : kNumBF16PerWarpBytes) / 32 * lane_id),
                            combined_values,
                            log_amax_buffers[stage_idx][division_idx],
                            log_amin_buffers[stage_idx][division_idx],
                            enable_cast,
                            topk_weight);
                    } else {
                        int tma_offset = kNumBF16PerWarpBytes * decode_warp_idx;
                        decode_and_accumulate<kNumRecvUnrolls>(
                            reinterpret_cast<uint32_t*>(tma_ld_buffers[stage_idx] + tma_offset + kNumBF16PerWarpBytes / 32 * lane_id),
                            combined_values,
                            0,
                            0,
                            false,
                            topk_weight);
                    }

                    if (elect_one_sync())
                        mbarrier_arrive(empty_barriers[stage_idx]);
                    stage_idx = (stage_idx + 1) % kNumStages;
                }
                tma_store_wait<0>();

                #pragma unroll
                for (int k = 0; k < kNumRecvUnrolls * 4; ++k) {
                    auto combined_pack = __hip_bfloat162(combined_values[k * 2], combined_values[k * 2 + 1]);
                    tma_st_buffers[decode_warp_idx][kNumRecvUnrolls * 4 * lane_id + k] = *reinterpret_cast<uint32_t*>(&combined_pack);
                }
                tma_store_fence();
                if (elect_one_sync()) {
                    tma_store_1d(tma_st_buffers[decode_warp_idx],
                                 static_cast<int4*>(combined_x) + token_idx * hidden_bf16_int4 + decode_warp_idx * kNumRecvUnrolls * 32,
                                 kNumBF16PerWarpBytes);
                }
                __syncwarp();
            }
        }
    }
}

void combine(void* combined_x,
             void* rdma_recv_x,
             int* rdma_recv_flag,
             void* rdma_send_x,
             const void* x,
             const topk_idx_t* topk_idx,
             const float* topk_weights,
             const int* src_info,
             const int64_t* layout_range,
             int* mask_buffer_ptr,
             int64_t* combine_wait_recv_cost_stats,
             int* next_clean,
             int num_next_clean_int,
             int num_combined_tokens,
             int hidden,
             int num_max_dispatch_tokens_per_rank,
             int num_topk,
             int num_experts,
             int rank,
             int num_ranks,
             bool use_logfmt,
             void* workspace,
             int num_device_sms,
             hipStream_t stream,
             int phases,
             bool zero_copy) {
    constexpr int kNumMaxTopk = 11;
    const int num_warp_groups = ceil_div(num_experts, num_device_sms);
    const int num_warps_per_group = 32 / num_warp_groups;
    const int num_recv_per_sm = ceil_div(num_combined_tokens, num_device_sms);
    EP_HOST_ASSERT(num_warp_groups > 0 and num_warps_per_group > 0 and num_recv_per_sm >= 0);

    const auto num_warps = num_warp_groups * num_warps_per_group;
    const auto num_sms =
        max(ceil_div(num_experts, num_warp_groups), num_recv_per_sm == 0 ? 1 : ceil_div(num_combined_tokens, num_recv_per_sm));

    // Check workspace
    auto atomic_clean_flag = static_cast<int*>(workspace);
    EP_HOST_ASSERT(sizeof(int) <= NUM_WORKSPACE_BYTES);
    EP_HOST_ASSERT(num_topk <= kNumMaxTopk);

    // Online cast cannot use zero-copy
    EP_HOST_ASSERT(not(zero_copy and use_logfmt));

    constexpr int kNumStages = 3;
    constexpr int kNumMaxUnrolls = 4;
    constexpr int kMaxNumGroups = 2;

    // Send buffer size
    const int num_meta_bytes = hidden / 128 * 4;
    const int num_send_tma_bytes = 32 * sizeof(int4) * kNumMaxUnrolls + 16;
    const int smem_send_size = num_warps * (kNumStages * num_send_tma_bytes + num_meta_bytes);

    // Receive buffer size
    const int num_recv_tma_bytes = 16 + hidden * 2;
    const int smem_recv_size = kMaxNumGroups * (kNumStages * num_recv_tma_bytes + hidden * 2 + kNumStages * num_meta_bytes * 3);

    // Total requirement
    const int smem_size = max(smem_send_size, smem_recv_size);

#define COMBINE_LAUNCH_CASE(hidden)                                                                                                \
    {                                                                                                                              \
        auto combine_func =                                                                                                        \
            use_logfmt ? combine<true, hidden, kNumMaxTopk, kNumMaxUnrolls> : combine<false, hidden, kNumMaxTopk, kNumMaxUnrolls>; \
        SET_SHARED_MEMORY_FOR_TMA(combine_func);                                                                                   \
        LAUNCH_KERNEL(&cfg,                                                                                                        \
                      combine_func,                                                                                                \
                      combined_x,                                                                                                  \
                      rdma_recv_x,                                                                                                 \
                      rdma_recv_flag,                                                                                              \
                      rdma_send_x,                                                                                                 \
                      x,                                                                                                           \
                      topk_idx,                                                                                                    \
                      topk_weights,                                                                                                \
                      src_info,                                                                                                    \
                      layout_range,                                                                                                \
                      mask_buffer_ptr,                                                                                             \
                      combine_wait_recv_cost_stats,                                                                                \
                      next_clean,                                                                                                  \
                      num_next_clean_int,                                                                                          \
                      atomic_clean_flag,                                                                                           \
                      num_combined_tokens,                                                                                         \
                      hidden,                                                                                                      \
                      num_topk,                                                                                                    \
                      num_max_dispatch_tokens_per_rank,                                                                            \
                      num_experts,                                                                                                 \
                      rank,                                                                                                        \
                      num_ranks,                                                                                                   \
                      num_warp_groups,                                                                                             \
                      num_warps_per_group,                                                                                         \
                      phases,                                                                                                      \
                      zero_copy);                                                                                                  \
    }                                                                                                                              \
    break

    SETUP_LAUNCH_CONFIG(num_sms, num_warps * 32, stream);
    SWITCH_HIDDEN(COMBINE_LAUNCH_CASE);
#undef COMBINE_LAUNCH_CASE
}

template <int kNumThreads>
__launch_bounds__(kNumThreads, 1) __global__ void query_mask_buffer(int* mask_buffer_ptr, int num_ranks, int* mask_tensor) {
    const auto num_sms = static_cast<int>(gridDim.x);
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto num_threads = num_sms * kNumThreads;
    const auto thread_id = sm_id * kNumThreads + static_cast<int>(threadIdx.x);
    for (int rank_id = thread_id; rank_id < num_ranks; rank_id += num_threads) {
        mask_tensor[rank_id] = mask_buffer_ptr[rank_id];
    }
}

void query_mask_buffer(int* mask_buffer_ptr, int num_ranks, int* mask_tensor, hipStream_t stream) {
    constexpr int num_sms = 1;
    constexpr int kNumThreads = 1024;
    SETUP_LAUNCH_CONFIG(num_sms, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, query_mask_buffer<kNumThreads>, mask_buffer_ptr, num_ranks, mask_tensor);
}

template <int kNumThreads>
__launch_bounds__(kNumThreads, 1) __global__ void update_mask_buffer(int* mask_buffer_ptr, int rank_to_mask, bool mask) {
    const auto sm_id = static_cast<int>(blockIdx.x);
    const auto thread_id = static_cast<int>(threadIdx.x);
    if (sm_id == 0 && thread_id == 0) {
        atomicExch(mask_buffer_ptr + rank_to_mask, mask ? 1 : 0);
    }
}

void update_mask_buffer(int* mask_buffer_ptr, int rank, bool mask, hipStream_t stream) {
    constexpr int num_sms = 1;
    constexpr int kNumThreads = 32;
    SETUP_LAUNCH_CONFIG(num_sms, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, update_mask_buffer<kNumThreads>, mask_buffer_ptr, rank, mask);
}

template <int kNumThreads>
__launch_bounds__(kNumThreads, 1) __global__ void clean_mask_buffer(int* mask_buffer_ptr, int num_ranks) {
    auto thread_id = static_cast<int>(threadIdx.x);
    #pragma unroll
    for (int i = thread_id; i < num_ranks; i += kNumThreads)
        mask_buffer_ptr[i] = 0;
}

void clean_mask_buffer(int* mask_buffer_ptr, int num_ranks, hipStream_t stream) {
    constexpr int num_sms = 1;
    constexpr int kNumThreads = 32;
    SETUP_LAUNCH_CONFIG(num_sms, kNumThreads, stream);
    LAUNCH_KERNEL(&cfg, clean_mask_buffer<kNumThreads>, mask_buffer_ptr, num_ranks);
}

}  // namespace internode_ll

}  // namespace deep_ep
